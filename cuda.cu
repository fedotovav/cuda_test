#include "hip/hip_runtime.h"
#include "base.h"

int get_cuda_cores_num()
{
   int dev_id = 0;

   hipDeviceProp_t dev_prop;
   hipError_t error = hipGetDevice(&dev_id);
   error = hipGetDeviceProperties(&dev_prop, dev_id);

   int cores = 0;
   int mp = dev_prop.multiProcessorCount;

   switch (dev_prop.major)
   {
      case 2: // Fermi
         if (dev_prop.minor == 1)
            cores = mp * 48;
         else
            cores = mp * 32;

         break;

      case 3: // Kepler
         cores = mp * 192;
         break;

      case 5: // Maxwell
         cores = mp * 128;
         break;

      default:
         cout << "Unknown device type" << endl;
         break;
   }
   
   return cores;
}

__global__ void warm_up_kernel( double * a, double b )
{
   a[0] = b;
}

void warm_up()
{
   double * dev, host;

   double b = 26;

   hipError_t err;

   err = hipMalloc((void **)&dev, sizeof(double));
   err = hipMemcpy(dev, &host, sizeof(double), hipMemcpyHostToDevice);

   warm_up_kernel<<< dim3(1), dim3(1) >>>(dev, b);

   hipDeviceSynchronize();

   err = hipMemcpy(&host, dev, sizeof(double), hipMemcpyDeviceToHost);

   hipFree(dev);
}

template<typename T>
__global__ void calc_kernel( unsigned int op_num, T * res, T number )
{
   res[0] = 10e-10;

   for (unsigned int i = 0; i < op_num; ++i)
      res[0] *= number;
}

template<typename T>
__global__ void calc_kernel_parallel( unsigned int op_num, T * res, T number, int op_num_per_thread )
{
   int id = blockIdx.x * blockDim.x + threadIdx.x;

   if (id < op_num)
      for (unsigned int i = 0; i < op_num_per_thread; ++i)
         res[0] *= number;
}

template<typename T>
__global__ void calc_kernel_parallel_no_optimize( unsigned int op_num, T * res, T number, unsigned int rand_idx, int op_num_per_thread )
{
   int id = blockIdx.x * blockDim.x + threadIdx.x;

   if (id < op_num)
      for (unsigned int i = 0; i < op_num_per_thread; ++i)
         res[rand_idx] *= number;
}

template< typename T >
double dev_mem_alloc( T * vec, size_t size, T ** vec_dev )
{
   hipEvent_t startEvent, stopEvent;

   hipEventCreate(&startEvent);
   hipEventCreate(&stopEvent);

   hipEventRecord(startEvent, 0);

   hipError_t error = hipMalloc((void **)vec_dev, sizeof(T) * size);
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   error = hipMemcpy(*vec_dev, vec, sizeof(T) * size, hipMemcpyHostToDevice);
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   hipEventRecord(stopEvent, 0);
   hipEventSynchronize(stopEvent);

   float duration;
   hipEventElapsedTime(&duration, startEvent, stopEvent);

   hipEventDestroy(startEvent);
   hipEventDestroy(stopEvent);

   return duration;
}

template<typename T>
double get_solution_from_device( T * vec, size_t size, T ** vec_dev )
{
   hipError_t error;

   hipEvent_t startEvent, stopEvent;

   hipEventCreate(&startEvent);
   hipEventCreate(&stopEvent);

   hipEventRecord(startEvent, 0);

   error = hipMemcpy(vec, *vec_dev, sizeof(T) * size, hipMemcpyDeviceToHost);
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   hipFree(*vec_dev);

   hipEventRecord(stopEvent, 0);
   hipEventSynchronize(stopEvent);

   float duration;
   hipEventElapsedTime(&duration, startEvent, stopEvent);

   hipEventDestroy(startEvent);
   hipEventDestroy(stopEvent);

   return duration;
}

template< typename T >
double dev_partial_mem_alloc( T * vec, size_t size, T ** vec_dev )
{
   hipEvent_t startEvent, stopEvent;

   hipEventCreate(&startEvent);
   hipEventCreate(&stopEvent);

   hipEventRecord(startEvent, 0);

   hipError_t error = hipMalloc((void **)vec_dev, sizeof(T) * size);
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   error = hipMemcpy(*vec_dev, vec, sizeof(T) * size, hipMemcpyHostToDevice);
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   hipEventRecord(stopEvent, 0);
   hipEventSynchronize(stopEvent);

   float duration;
   hipEventElapsedTime(&duration, startEvent, stopEvent);

   hipEventDestroy(startEvent);
   hipEventDestroy(stopEvent);

   return duration;
}

template<typename T>
double get_partial_solution_from_device( T * vec, size_t size, T ** vec_dev )
{
   time_res_t time;

   hipError_t error;

   hipEvent_t startEvent, stopEvent;

   hipEventCreate(&startEvent);
   hipEventCreate(&stopEvent);

   hipEventRecord(startEvent, 0);

   error = hipMemcpy(vec, *vec_dev, sizeof(T) * size, hipMemcpyDeviceToHost);
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   hipFree(*vec_dev);

   hipEventRecord(stopEvent, 0);
   hipEventSynchronize(stopEvent);

   float duration;
   hipEventElapsedTime(&duration, startEvent, stopEvent);

   hipEventDestroy(startEvent);
   hipEventDestroy(stopEvent);

   return duration;
}

template<typename T>
double partial_mem_test( T * vec, size_t from, size_t to )
{
   warm_up();

   time_res_t time;

   T * vec_dev;

   double duration = 0;

   time.measure_start();

   size_t size = to - from;

   T * data = new T[size];

   for (size_t i = 0; i < size; ++i)
      data[i] = vec[i + from];

   time.gpu_mem_partial_time_ = time.measure_finish();

   duration += dev_partial_mem_alloc(vec, size, &vec_dev);

   duration += get_partial_solution_from_device(vec, size, &vec_dev);

   time.measure_start();

   for (size_t i = 0; i < size; ++i)
      vec[i + from] = data[i];

   delete[] data;

   time.gpu_mem_partial_time_ += time.measure_finish() + duration;

   return time.gpu_mem_partial_time_;
}

template<typename T>
double mem_test( T * vec, size_t size )
{
   warm_up();

   double time = 0;

   T * vec_dev;

   time += dev_mem_alloc(vec, size, &vec_dev);

   time += get_solution_from_device(vec, size, &vec_dev);

   return time;
}

template<typename T>
double calc( T * vec, size_t op_num )
{
   warm_up();

   time_res_t time;

   hipError_t error;

   hipEvent_t startEvent, stopEvent;

   float duration;

   T * dev, host;

   T b = 10e-10;

   error = hipMalloc((void **)&dev, sizeof(double));
   error = hipMemcpy(dev, &host, sizeof(double), hipMemcpyHostToDevice);

   hipEventCreate(&startEvent);
   hipEventCreate(&stopEvent);

   hipEventRecord(startEvent, 0);

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   calc_kernel<T><<<1, 1>>>(op_num, dev, b);

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   hipDeviceSynchronize();

   hipEventRecord(stopEvent, 0);
   hipEventSynchronize(stopEvent);

   hipEventElapsedTime(&duration, startEvent, stopEvent);

   hipFree(dev);

   hipEventDestroy(startEvent);
   hipEventDestroy(stopEvent);

   return duration;
}

template<typename T>
double calc_parallel( T * vec, size_t op_num, size_t threads_num )
{
   if (!threads_num)
      return 0;

   hipEvent_t startEvent, stopEvent;

   time_res_t time;

   T * dev, host;

   T b = 10e-10;

   hipError_t error;

   error = hipMalloc((void **)&dev, sizeof(double));
   error = hipMemcpy(dev, &host, sizeof(double), hipMemcpyHostToDevice);

   float duration;

   int block_dim = (threads_num > 1024) ? 1024 : threads_num;
   int grid_dim = (int)ceil(threads_num / 1024);
   int op_num_per_thread = op_num / threads_num;

   if (!grid_dim)
      grid_dim = 1;

   hipEventCreate(&startEvent);
   hipEventCreate(&stopEvent);

   hipEventRecord(startEvent, 0);

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   calc_kernel_parallel<T><<<grid_dim, block_dim>>>(op_num, dev, b, op_num_per_thread);

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   hipDeviceSynchronize();

   hipEventRecord(stopEvent, 0);
   hipEventSynchronize(stopEvent);

   hipEventElapsedTime(&duration, startEvent, stopEvent);

   hipEventDestroy(startEvent);
   hipEventDestroy(stopEvent);

   return duration;
}

template<typename T>
double calc_parallel_no_optimize( T * vec, size_t size, size_t op_num, size_t threads_num )
{
   if (!threads_num)
      return 0;

   hipEvent_t startEvent, stopEvent;

   time_res_t time;

   T b = 10e-10;

   hipError_t error;

   T * vec_dev;

   unsigned int rand_idx = (size - 1) * (double)rand() / (double)RAND_MAX;

   dev_mem_alloc(vec, size, &vec_dev);

   float duration;

   int block_dim = (threads_num > 1024) ? 1024 : threads_num;
   int grid_dim = (int)ceil(threads_num / 1024);
   int op_num_per_thread = op_num / threads_num;

   if (!grid_dim)
      grid_dim = 1;

   hipEventCreate(&startEvent);
   hipEventCreate(&stopEvent);

   hipEventRecord(startEvent, 0);

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   calc_kernel_parallel_no_optimize<T><<<grid_dim, block_dim>>>(op_num, vec_dev, b, rand_idx, op_num_per_thread);

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error: %s, line(%d)\n", hipGetErrorString(error), __LINE__);

   hipDeviceSynchronize();

   hipEventRecord(stopEvent, 0);
   hipEventSynchronize(stopEvent);

   hipEventElapsedTime(&duration, startEvent, stopEvent);

   hipEventDestroy(startEvent);
   hipEventDestroy(stopEvent);

   get_solution_from_device(vec, size, &vec_dev);

   return duration;
}

template< typename T >
double measure_mem_time( T * vec, size_t size )
{
   int devID = 0;

   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);

   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited)
   {
      fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
      return 0;
   }

   if (error != hipSuccess) printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);

   double duration = mem_test(vec, size);

   hipDeviceReset();

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error code (%d) at line(%d): %s\n", error, __LINE__, hipGetErrorString(error));

   return duration;
}

template< typename T >
double measure_partial_mem_time( T * vec, size_t from, size_t to )
{
   int devID = 0;

   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);

   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited)
   {
      fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
      return 0;
   }

   if (error != hipSuccess) printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);

   double duration = partial_mem_test(vec, from, to);

   hipDeviceReset();

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error code (%d) at line(%d): %s\n", error, __LINE__, hipGetErrorString(error));

   return duration;
}

template< typename T >
double measure_calc_time( T * vec, size_t op_num )
{
   int devID = 0;

   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);

   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited)
   {
      fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
      return 0;
   }

   if (error != hipSuccess) printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);

   double duration = calc<T>(vec, op_num);

   hipDeviceReset();

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error code (%d) at line(%d): %s\n", error, __LINE__, hipGetErrorString(error));

   return duration;
}

template< typename T >
double measure_parallel_calc_time( T * vec, size_t op_num, size_t thread_num )
{
   int devID = 0;

   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);

   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited)
   {
      fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
      return 0;
   }

   if (error != hipSuccess) printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);

   double duration = calc_parallel<T>(vec, op_num, thread_num);

   hipDeviceReset();

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error code (%d) at line(%d): %s\n", error, __LINE__, hipGetErrorString(error));

   return duration;
}

template< typename T >
double measure_parallel_calc_no_optimize_time( T * vec, size_t size, size_t op_num, size_t thread_num )
{
   int devID = 0;

   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);

   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited)
   {
      fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
      return 0;
   }

   if (error != hipSuccess) printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);

   double duration = calc_parallel_no_optimize<T>(vec, size, op_num, thread_num);

   hipDeviceReset();

   error = hipPeekAtLastError();
   if (error != hipSuccess) printf("CUDA error code (%d) at line(%d): %s\n", error, __LINE__, hipGetErrorString(error));

   return duration;
}

double measure_mem_gpu( int * vec, size_t size )
{
   return measure_mem_time(vec, size);
}

double measure_mem_gpu( float * vec, size_t size )
{
   return measure_mem_time(vec, size);
}

double measure_mem_gpu( double * vec, size_t size )
{
   return measure_mem_time(vec, size);
}

double measure_mem_gpu( unsigned int * vec, size_t size )
{
   return measure_mem_time(vec, size);
}

double measure_partial_mem_gpu( int * vec, size_t from, size_t to )
{
   return measure_partial_mem_time(vec, from, to);
}

double measure_partial_mem_gpu( float * vec, size_t from, size_t to )
{
   return measure_partial_mem_time(vec, from, to);
}

double measure_partial_mem_gpu( double * vec, size_t from, size_t to )
{
   return measure_partial_mem_time(vec, from, to);
}

double measure_partial_mem_gpu( unsigned int * vec, size_t from, size_t to )
{
   return measure_partial_mem_time(vec, from, to);
}

double measure_calc_time_gpu( int * vec, size_t op_num )
{
   return measure_calc_time(vec, op_num);
}

double measure_calc_time_gpu( float * vec, size_t op_num )
{
   return measure_calc_time(vec, op_num);
}

double measure_calc_time_gpu( double * vec, size_t op_num )
{
   return measure_calc_time(vec, op_num);
}

double measure_calc_time_gpu( unsigned int * vec, size_t op_num )
{
   return measure_calc_time(vec, op_num);
}

double measure_parallel_calc_time_gpu( int * vec, size_t op_num, size_t thread_num )
{
   return measure_parallel_calc_time(vec, op_num, thread_num);
}

double measure_parallel_calc_time_gpu( float * vec, size_t op_num, size_t thread_num )
{
   return measure_parallel_calc_time(vec, op_num, thread_num);
}

double measure_parallel_calc_time_gpu( double * vec, size_t op_num, size_t thread_num )
{
   return measure_parallel_calc_time(vec, op_num, thread_num);
}

double measure_parallel_calc_time_gpu( unsigned int * vec, size_t op_num, size_t thread_num )
{
   return measure_parallel_calc_time(vec, op_num, thread_num);
}

double measure_parallel_calc_no_optimize_time_gpu( int * vec, size_t size, size_t op_num, size_t thread_num )
{
   return measure_parallel_calc_no_optimize_time(vec, size, op_num, thread_num);
}

double measure_parallel_calc_no_optimize_time_gpu( float * vec, size_t size, size_t op_num, size_t thread_num )
{
   return measure_parallel_calc_no_optimize_time(vec, size, op_num, thread_num);
}

double measure_parallel_calc_no_optimize_time_gpu( double * vec, size_t size, size_t op_num, size_t thread_num )
{
   return measure_parallel_calc_no_optimize_time(vec, size, op_num, thread_num);
}

double measure_parallel_calc_no_optimize_time_gpu( unsigned int * vec, size_t size, size_t op_num, size_t thread_num )
{
   return measure_parallel_calc_no_optimize_time(vec, size, op_num, thread_num);
}
